#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


using namespace std;

__global__ void blur_kernel(float* d_input, int width, int height, float* d_output, float * d_msk,   int m_size){

    int threadIdPixel, blockId;
    int neighbourPixel;
    float val_pixel = 0;

    blockId         = (gridDim.x * blockIdx.y) + blockIdx.x;
    threadIdPixel   = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

    for (int i = 0; i < m_size ; i++){
        for (int j = 0; j < m_size ; j++){
            neighbourPixel =threadIdPixel + (j- m_size/2) +(i-m_size/2)*width ;                 
            if(neighbourPixel >= 0 && neighbourPixel < width * height ){
                val_pixel = val_pixel +  d_input[neighbourPixel] * d_msk[i*m_size+j];
            }
        }
    }
    if (threadIdPixel <= width * height )
        d_output[threadIdPixel] = val_pixel;
}




void blur_gpu(float * image_in, int width, int height, float * image_out,  float mask[], int m_size, int threadPerBlockx, int threadPerBlocky){
    
    // Reservar memoria en la GPU
    float *d_img_in; float *d_img_out; float *d_mask;
    int nbx;//Número de blques x
    int nby;//Número de blques Y
    unsigned int size_img = width * height * sizeof(float);
    unsigned int size_msk = m_size * m_size * sizeof(int);

    width % threadPerBlockx == 0 ? nbx = width / threadPerBlockx : nbx = width / threadPerBlockx + 1;
    height % threadPerBlocky == 0 ? nby = height / threadPerBlocky : nby = height / threadPerBlocky + 1;

    // Inicializo variables para medir tiempos
    CLK_CUEVTS_INIT;
    CLK_POSIX_INIT;


    CLK_CUEVTS_START;
    CLK_POSIX_START;
    CUDA_CHK(hipMalloc( (void**)&d_img_in   , size_img));//Reservo memoria en el device para la imagen original
    CUDA_CHK(hipMalloc( (void**)&d_img_out  , size_img));//Reservo memoria en el device para la imagen de salida
    CUDA_CHK(hipMalloc( (void**)&d_mask     , size_msk));//Reservo memoria para la mascada
    CLK_POSIX_STOP;
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    CLK_POSIX_ELAPSED;
    float t_elap_cuda_malloc = t_elap_cuda;
    float t_elap_get_malloc = t_elap_get;
    

    
    // copiar imagen y máscara a la GPU
    CLK_POSIX_START;
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(d_img_in  , image_in  , size_img, hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy(d_img_out , image_out , size_img, hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy(d_mask    , &mask[0]  , size_msk, hipMemcpyHostToDevice));
    CLK_CUEVTS_STOP;
    CLK_POSIX_STOP;
    CLK_CUEVTS_ELAPSED;
    CLK_POSIX_ELAPSED;
    float t_elap_cuda_cpyHtoD = t_elap_cuda;
    float t_elap_get_cpyHtoD = t_elap_get;

    // configurar grilla y lanzar kernel
    dim3 grid(nbx,nby)  ;
    dim3 block(threadPerBlockx,threadPerBlocky) ;

    CLK_POSIX_START;
    CLK_CUEVTS_START;
    blur_kernel <<< grid, block >>> (d_img_in, width, height, d_img_out, d_mask,  m_size); 
    CLK_CUEVTS_STOP;
    
    // Obtengo los posibles errores en la llamada al kernel
	CUDA_CHK(hipGetLastError());

	// Obligo al Kernel a llegar al final de su ejecucion y hacer obtener los posibles errores
	CUDA_CHK(hipDeviceSynchronize());

    CLK_POSIX_STOP;
    CLK_CUEVTS_ELAPSED;
    CLK_POSIX_ELAPSED;
    float t_elap_cuda_kernel = t_elap_cuda;
    float t_elap_get_kernel = t_elap_get;

    // transferir resultado a la memoria principal
    CLK_POSIX_START;
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(image_out  , d_img_out , size_img, hipMemcpyDeviceToHost));
    CLK_CUEVTS_STOP;
    CLK_POSIX_STOP;
    CLK_CUEVTS_ELAPSED;
    CLK_POSIX_ELAPSED;
    float t_elap_cuda_cpyDtoH = t_elap_cuda;
    float t_elap_get_cpyDtoH = t_elap_get;
	
    // liberar la memoria
    CLK_POSIX_START;
    CLK_CUEVTS_START;
    hipFree(d_img_in); hipFree(d_img_out) ; hipFree(d_mask);
    CLK_CUEVTS_STOP;
    CLK_POSIX_STOP;
    CLK_CUEVTS_ELAPSED;
    CLK_POSIX_ELAPSED;
    float t_elap_cuda_free = t_elap_cuda;
    float t_elap_get_free = t_elap_get;

    printf("blur filter timing GPU:\n");
    printf("type:     | cudaEvents      | gettimeofday\n");
    printf("malloc:   | %06.3f ms       | %06.3f ms\n", t_elap_cuda_malloc, t_elap_get_malloc);
    printf("cpyHtoD:  | %06.3f ms       | %06.3f ms\n", t_elap_cuda_cpyHtoD, t_elap_get_cpyHtoD);
    printf("kernel:   | %06.3f ms       | %06.3f ms\n", t_elap_cuda_kernel, t_elap_get_kernel);
    printf("cpyDtoH:  | %06.3f ms       | %06.3f ms\n", t_elap_cuda_cpyDtoH, t_elap_get_cpyDtoH);
    printf("free:     | %06.3f ms       | %06.3f ms\n", t_elap_cuda_free, t_elap_get_free);
    printf("TOTAL:    | %06.3f ms       | %06.3f ms\n", t_elap_cuda_malloc + t_elap_cuda_cpyHtoD + t_elap_cuda_kernel + t_elap_cuda_cpyDtoH + t_elap_cuda_free + t_elap_cuda_malloc, t_elap_get_malloc + t_elap_get_cpyHtoD + t_elap_get_kernel + t_elap_get_cpyDtoH + t_elap_get_free + t_elap_get_malloc);
    printf("\n");
}

void blur_cpu(float * img_in, int width, int height, float * img_out, float msk[], int m_size){

    float val_pixel=0;

    // Inicializo variables para medir tiempos
    CLK_POSIX_INIT;
    
    CLK_POSIX_START;
    //para cada pixel aplicamos el filtro
    for(int imgx=0; imgx < width ; imgx++){
        for(int imgy=0; imgy < height; imgy++){

            val_pixel = 0;

            // aca aplicamos la mascara
            for (int i = 0; i < m_size ; i++){
                for (int j = 0; j < m_size ; j++){
                    
                    int ix =imgx + i - m_size/2;
                    int iy =imgy + j - m_size/2;
                    
                    if(ix >= 0 && ix < width && iy>= 0 && iy < height )
                        val_pixel = val_pixel +  img_in[iy * width +ix] * msk[i*m_size+j];
                    }
            }      
            // guardo valor resultado
            img_out[imgy*width+imgx]= val_pixel;
        }
    }
    CLK_POSIX_STOP;
    CLK_POSIX_ELAPSED;

    float t_elap = t_elap_get;

    printf("blur filter timing CPU:\n");
    printf("type:                       | gettimeofday\n");
    printf("TOTAL:                      | %06.3f ms\n",t_elap);
    printf("\n");
}
